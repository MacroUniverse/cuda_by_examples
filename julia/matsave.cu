#include "hip/hip_runtime.h"
#include "matsave.h"

using namespace std;

#ifdef MATFILE_BINARY

// matsave()

void matsave(const Uchar s, const std::string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateNumericMatrix(1, 1, mxUINT8_CLASS, mxREAL);
	Uchar *pps = (Uchar*)mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Int s, const std::string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
	Int *pps = (Int*)mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Doub s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateDoubleMatrix(1, 1, mxREAL);
	auto pps = mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Complex s, const string &varname, MATFile *pfile)
{
	mxArray *pa;
	pa = mxCreateDoubleMatrix(1, 1, mxCOMPLEX);

	auto ppa = mxGetPr(pa);
	ppa[0] = real(s);
	ppa = mxGetPi(pa);
	ppa[0] = imag(s);
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(VecUchar_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxUINT8_CLASS, mxREAL);
	Uchar *ppv = (Uchar*)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecInt_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxINT32_CLASS, mxREAL);
	Int *ppv = (Int*)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecDoub_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxDOUBLE_CLASS, mxREAL);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecComplex_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	string str;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxDOUBLE_CLASS, mxCOMPLEX);
	auto ppvr = mxGetPr(pv);
	auto ppvi = mxGetPi(pv);
	for (i = 0; i < n; ++i) {
		ppvr[i] = real(v[i]);
		ppvi[i] = imag(v[i]);
	}
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(MatUchar_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxUINT8_CLASS, mxREAL);
		Uchar *ppa = (Uchar*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[step1*i][step2*j];
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxUINT8_CLASS, mxREAL);
		Uchar *ppa = (Uchar*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[i][j];
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatInt_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxINT32_CLASS, mxREAL);
		Int *ppa = (Int*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[step1*i][step2*j];
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxINT32_CLASS, mxREAL);
		Int *ppa = (Int*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[i][j];
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatDoub_I &a, const string &varname, MATFile *pfile, 
			const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m*j] = a[step1*i][step2*j];
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m*j] = a[i][j];
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatComplex_I &a, const string &varname, MATFile *pfile,
			const Int step1, const Int step2)
{
	Int i, j, m, n, ind;
	Complex c;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1)/step1; n = (a.ncols() + step2 - 1)/step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa);
		auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ind = i + m * j; c = a[step1*i][step2*j];
				ppar[ind] = real(c);
				ppai[ind] = imag(c);
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa);
		auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ind = i + m * j; c = a[i][j];
				ppar[ind] = real(c);
				ppai[ind] = imag(c);
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DDoub_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q, mn;
	mxArray *pa;
	m = a.dim1(); n = a.dim2(); q = a.dim3(); mn = m * n;
	if (step1 > 1 || step2 > 1 || step3 > 1) {
		m = (m + step1 - 1) / step1; n = (n + step2 - 1) / step2;
		q = (q + step3 - 1) / step3;
		size_t sz[3]{ (size_t)m,(size_t)n,(size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k)
					ppa[i + m * j + mn * k] = a[step1*i][step2*j][step3*k];
	}
	else {
		size_t sz[3]{ (size_t)m,(size_t)n,(size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k)
					ppa[i + m * j + mn * k] = a[i][j][k];
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

/* specify xyz = 'x','y' or 'z', and take Nslice at indslice[i]
if xyz = 'x', step1 is in y direction, step2 is in z direction, save pa[iy][iz][ix].
if xyz = 'y', step1 is in z direction, step2 is in x direction, save pa[iz][ix][iy].
if xyz = 'z', step1 is in x direction, step2 is in y direction, save pa[ix][iy][iz]. */
void matsave(Mat3DDoub_I &a, const std::string &varname, MATFile *pfile,
			const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, mn, Nslice{ slice.size() };
	mxArray *pa;
	if (xyz == 'x') {
		Int ind;
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (j = 0; j < m; ++j)
				for (k = 0; k < n; ++k)
					ppa[j + m*k + mn*i] = a[ind][step1*j][step2*k];
		}
	}
	else if (xyz == 'y') {
		Int ind;
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (k = 0; k < m; ++k)
				for (i = 0; i < n; ++i)
					ppa[k + m*i + mn*j] = a[step2*i][ind][step1*k];
		}
	}
	else if (xyz == 'z') {
		Int ind;
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (i = 0; i < m; ++i)
				for (j = 0; j < n; ++j)
					ppa[i + m*j + mn*k] = a[step1*i][step2*j][ind];
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DComplex_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q, mn, ind;
	mxArray *pa;
	Complex c;
	m = a.dim1(); n = a.dim2(); q = a.dim3(); mn = m * n;
	if (step1 > 1 || step2 > 1 || step3 > 1) {
		m = (m + step1 - 1) / step1; n = (n + step2 - 1) / step2;
		q = (q + step3 - 1) / step3;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k) {
					ind = i + m * j + mn * k;
					c = a[step1*i][step2*j][step3*k];
					ppar[ind] = real(c); ppai[ind] = imag(c);
				}
	}
	else {
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k) {
					ind = i + m * j + mn * k;
					c = a[i][j][k];
					ppar[ind] = real(c); ppai[ind] = imag(c);
				}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

/* specify xyz = 'x','y' or 'z', and take Nslice at indslice[i]
if xyz = 'x', step1 is in y direction, step2 is in z direction, save pa[iy][iz][ix].
if xyz = 'y', step1 is in z direction, step2 is in x direction, save pa[iz][ix][iy].
if xyz = 'z', step1 is in x direction, step2 is in y direction, save pa[ix][iy][iz]. */
void matsave(Mat3DComplex_I &a, const std::string &varname, MATFile *pfile,
			const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{	
	Int i, j, k, m, n, mn, inda, Nslice{ slice.size() };
	mxArray *pa;
	Complex c;
	if (xyz == 'x') {
		Int ind;
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (j = 0; j < m; ++j)
				for (k = 0; k < n; ++k) {
					inda = j + m * k + mn * i;
					c = a[ind][step1*j][step2*k];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else if (xyz == 'y') {
		Int ind;
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (k = 0; k < m; ++k)
				for (i = 0; i < n; ++i) {
					inda = k + m * i + mn * j;
					c = a[step2*i][ind][step1*k];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else if (xyz == 'z') {
		Int ind;
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (i = 0; i < m; ++i)
				for (j = 0; j < n; ++j) {
					inda = i + m * j + mn * k;
					c = a[step1*i][step2*j][ind];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

// matload()

void matload(Int &i, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	auto pps = mxGetPr(ps);
	i = (Int)pps[0];
	mxDestroyArray(ps);
}

void matload(Doub &s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	auto pps = mxGetPr(ps);
	s = pps[0];
	mxDestroyArray(ps);
}

void matload(Complex &s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	auto ppsr = mxGetPr(ps);
	auto ppsi = mxGetPi(ps);
	if (ppsi)
		s = Complex(ppsr[0], ppsi[0]);
	else
		s = ppsr[0];
	mxDestroyArray(ps);
}

void matload(VecInt_O &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	n = (Int)mxGetDimensions(pv)[1];
	if (v.size() != n) v.resize(n);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = (Int)round(ppv[i]);
	mxDestroyArray(pv);
}

void matload(VecDoub_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	n = (Int)mxGetDimensions(pv)[1];
	if (v.size() != n) v.resize(n);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = ppv[i];
	mxDestroyArray(pv);
}

void matload(VecComplex_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	n = (Int)mxGetDimensions(pv)[1];
	v.resize(n);
	auto ppvr = mxGetPr(pv);
	auto ppvi = mxGetPi(pv);
	if (ppvi)
		for (i = 0; i < n; ++i)
			v[i] = Complex(ppvr[i], ppvi[i]);
	else
		for (i = 0; i < n; ++i)
			v[i] = ppvr[i];
	mxDestroyArray(pv);
}

void matload(MatInt_O &a, const std::string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = (Int)ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatDoub_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatComplex_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n, ind;
	mxArray *pa;
	pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppar = mxGetPr(pa);
	auto ppai = mxGetPi(pa);
	if (ppai)
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j){
			ind = i + m * j;
			a[i][j] = Complex(ppar[ind], ppai[ind]);
		}
	else
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppar[i + m*j];
	mxDestroyArray(pa);
}

void matload(Mat3DDoub_O &a, const std::string &varname, MATFile *pfile)
{
	Int i, j, k, m, n, q, mn;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1]; q = (Int)sz[2]; mn = m*n;
	a.resize(m, n, q);
	auto *ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
	for (j = 0; j < n; ++j)
	for (k = 0; k < q; ++k)
		a[i][j][k] = ppa[i + m*j + mn*k];
	mxDestroyArray(pa);
}

void matload(Mat3DComplex_O &a, const std::string &varname, MATFile *pfile)
{
	Int i, j, k, m, n, q, mn, ind;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1]; q = (Int)sz[2]; mn = m*n;
	a.resize(m, n, q);
	auto *ppar = mxGetPr(pa);
	auto *ppai = mxGetPi(pa);
	if (ppai)
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
		for (k = 0; k < q; ++k){
			ind = i + m*j + mn*k;
			a[i][j][k] = Complex(ppar[ind], ppai[ind]);
		}
	else
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
		for (k = 0; k < q; ++k)
			a[i][j][k] = ppar[i + m*j + mn*k];
	mxDestroyArray(pa);
}

#else /*#ifdef MATFILE_BINARY*/
// text mode

// change extension from ".mat" to
MATFile *matOpen(std::string fname, std::string rw)
{
	Int N = fname.size();
	fname += "t";
	MATFile* pfile = new ofstream;
	*pfile = ofstream(fname);
#ifdef MATFILE_PRECISION
	pfile->precision(MATFILE_PRECISION);
#endif
	return pfile;
}

void matClose(MATFile* pfile)
{
	pfile->close();
	delete pfile;
}

void matsave(const Uchar s, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 3 << '\n';
	// write dimension info
	*pfile << 0 << '\n';
	// write matrix data
	*pfile << (Int)s << '\n';
}

void matsave(const Int s, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 2 << '\n';
	// write dimension info
	*pfile << 0 << '\n';
	// write matrix data
	*pfile << s << '\n';
}

void matsave(const Doub s, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 0 << '\n';
	// write dimension info
	*pfile << 0 << '\n';
	// write matrix data
	*pfile << s << '\n';
}

void matsave(const Complex s, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 1 << '\n';
	// write dimension info
	*pfile << 0 << '\n';
	// write matrix data
	if (imag(s) == 0)
		*pfile << real(s) << '\n';
	else
		*pfile << real(s) << '+' << imag(s) << "i\n";
}

void matsave(VecUchar_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 3 << '\n';
	// write dimension info
	n = v.size();
	*pfile << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		*pfile << (Int)v[i] << '\n';
	}
}

void matsave(VecInt_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 2 << '\n';
	// write dimension info
	n = v.size();
	*pfile << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		*pfile << v[i] << '\n';
	}
}

void matsave(VecDoub_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 0 << '\n';
	// write dimension info
	n = v.size();
	*pfile << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		*pfile << v[i] << '\n';
	}
}

void matsave(VecComplex_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	Doub cr, ci;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 1 << '\n';
	// write dimension info
	n = v.size();
	*pfile << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		cr = real(v[i]); ci = imag(v[i]);
		if (ci == 0)
			*pfile << cr << '\n';
		else
			*pfile << cr << '+' << ci << "i\n";
	}
}

void matsave(MatUchar_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 3 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	*pfile << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			*pfile << (Int)a[step1*i][step2*j] << '\n';
		}
}

void matsave(MatInt_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 2 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	*pfile << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			*pfile << a[step1*i][step2*j] << '\n';
		}
}

void matsave(MatDoub_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 0 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	*pfile << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			*pfile << a[step1*i][step2*j] << '\n';
		}
}

void matsave(MatComplex_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	Complex c; Doub cr, ci;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 1 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	*pfile << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			c = a[step1*i][step2*j]; cr = real(c); ci = imag(c);
			if (ci == 0)
				*pfile << cr << '\n';
			else
				*pfile << cr << '+' << ci << "i\n";
		}
}

void matsave(Mat3DDoub_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 0 << '\n';
	// write dimension info
	m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
	q = (a.dim3() + step3 - 1) / step3;
	*pfile << 3 << '\n' << m << '\n' << n << '\n' << q << '\n';
	// write matrix data
	for (k = 0; k < q; ++k)
	for (j = 0; j < n; ++j)
	for (i = 0; i < m; ++i)
		*pfile << a[step1*i][step2*j][step3*k] << '\n';
}

/* specify xyz = 'x','y' or 'z', and take Nslice at indslice[i]
if xyz = 'x', step1 is in y direction, step2 is in z direction, save pa[iy][iz][ix].
if xyz = 'y', step1 is in z direction, step2 is in x direction, save pa[iz][ix][iy].
if xyz = 'z', step1 is in x direction, step2 is in y direction, save pa[ix][iy][iz]. */
void matsave(Mat3DDoub_I &a, const std::string &varname, MATFile *pfile,
	const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, ind, Nslice{ slice.size() };
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 0 << '\n';
	if (xyz == 'x') {
		// write dimension info
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2;
		*pfile << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (k = 0; k < n; ++k)
				for (j = 0; j < m; ++j)
					*pfile << a[ind][step1*j][step2*k] << '\n';
		}
	}
	else if (xyz == 'y') {
		// write dimension info
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2;
		*pfile << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (i = 0; i < n; ++i)
				for (k = 0; k < m; ++k)
					*pfile << a[step2*i][ind][step1*k] << '\n';
		}
	}
	else if (xyz == 'z') {
		// write dimension info
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
		*pfile << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (j = 0; j < n; ++j)
				for (i = 0; i < m; ++i)
					*pfile << a[step1*i][step2*j][ind] << '\n';
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
}

void matsave(Mat3DComplex_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q;
	Complex c; Doub cr, ci;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 1 << '\n';
	// write dimension info
	m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
	q = (a.dim3() + step3 - 1) / step3;
	*pfile << 3 << '\n' << m << '\n' << n << '\n' << q << '\n';
	// write matrix data
	for (k = 0; k < q; ++k)
		for (j = 0; j < n; ++j)
			for (i = 0; i < m; ++i) {
				c = a[step1*i][step2*j][step3*k]; cr = real(c); ci = imag(c);
				if (ci == 0)
					*pfile << cr << '\n';
				else
					*pfile << cr << '+' << ci << "i\n";
			}
}

/* specify xyz = 'x','y' or 'z', and take Nslice at indslice[i]
if xyz = 'x', step1 is in y direction, step2 is in z direction, save pa[iy][iz][ix].
if xyz = 'y', step1 is in z direction, step2 is in x direction, save pa[iz][ix][iy].
if xyz = 'z', step1 is in x direction, step2 is in y direction, save pa[ix][iy][iz]. */
void matsave(Mat3DComplex_I &a, const std::string &varname, MATFile *pfile,
	const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, ind, Nslice{ slice.size() };
	Complex c; Doub cr, ci;
	// write variable name info
	n = varname.size();
	*pfile << n << '\n';
	for (i = 0; i < n; ++i) {
		*pfile << (Int)varname.at(i) << '\n';
	}
	// write data type info
	*pfile << 1 << '\n';
	if (xyz == 'x') {
		// write dimension info
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2;
		*pfile << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (k = 0; k < n; ++k)
				for (j = 0; j < m; ++j)
				{
					c = a[ind][step1*j][step2*k]; cr = real(c); ci = imag(c);
					if (ci == 0)
						*pfile << cr << '\n';
					else
						*pfile << cr << '+' << ci << "i\n";
				}
					
		}
	}
	else if (xyz == 'y') {
		// write dimension info
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2;
		*pfile << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (i = 0; i < n; ++i)
				for (k = 0; k < m; ++k) {
					c = a[step2*i][ind][step1*k]; cr = real(c); ci = imag(c);
					if (ci == 0)
						*pfile << cr << '\n';
					else
						*pfile << cr << '+' << ci << "i\n";
				}
		}
	}
	else if (xyz == 'z') {
		// write dimension info
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
		*pfile << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (j = 0; j < n; ++j)
				for (i = 0; i < m; ++i) {
					c = a[step1*i][step2*j][ind]; cr = real(c); ci = imag(c);
					if (ci == 0)
						*pfile << cr << '\n';
					else
						*pfile << cr << '+' << ci << "i\n";
				}
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
}

// matread functions

// how to write matread() functions in c++
//ifstream fin("test.matt");
//Doub cr = 0, ci = 0, d3 = 0, d4 = 0;
//Uchar suchar;
//fin >> cr;
//suchar = fin.get();
//fin >> ci;
//suchar = fin.get();
//suchar = fin.get();
//fin >> d3;
//fin >> suchar;
//fin >> d4;
//fin >> suchar;
//fin.ignore(1000, '\n');
//fin.close();

#endif /*#ifdef MATFILE_BINARY*/