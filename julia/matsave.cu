#include "matsave.h"

using namespace std;

#if defined(MATFILE_BINARY) || defined(MATFILE_DUAL)

void matsave(const Uchar s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateNumericMatrix(1, 1, mxUINT8_CLASS, mxREAL);
	Uchar *pps = (Uchar*)mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Int s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
	Int *pps = (Int*)mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Doub s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateDoubleMatrix(1, 1, mxREAL);
	auto pps = mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Complex s, const string &varname, MATFile *pfile)
{
	mxArray *pa;
	pa = mxCreateDoubleMatrix(1, 1, mxCOMPLEX);

	auto ppa = mxGetPr(pa);
	ppa[0] = real(s);
	ppa = mxGetPi(pa);
	ppa[0] = imag(s);
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(VecUchar_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxUINT8_CLASS, mxREAL);
	Uchar *ppv = (Uchar*)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecInt_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxINT32_CLASS, mxREAL);
	Int *ppv = (Int*)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecDoub_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxDOUBLE_CLASS, mxREAL);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecComplex_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	string str;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxDOUBLE_CLASS, mxCOMPLEX);
	auto ppvr = mxGetPr(pv);
	auto ppvi = mxGetPi(pv);
	for (i = 0; i < n; ++i) {
		ppvr[i] = real(v[i]);
		ppvi[i] = imag(v[i]);
	}
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(MatUchar_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;

	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	pa = mxCreateUninitNumericMatrix(m, n, mxUINT8_CLASS, mxREAL);
	Uchar *ppa = (Uchar*)mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j) {
			ppa[i + m * j] = a[step1*i][step2*j];
		}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatInt_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxINT32_CLASS, mxREAL);
		Int *ppa = (Int*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[step1*i][step2*j];
			}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatDoub_I &a, const string &varname, MATFile *pfile, 
			const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxREAL);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j) {
			ppa[i + m*j] = a[step1*i][step2*j];
		}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatComplex_I &a, const string &varname, MATFile *pfile,
			const Int step1, const Int step2)
{
	Int i, j, m, n, ind;
	Complex c;
	mxArray *pa;
	m = (a.nrows() + step1 - 1)/step1; n = (a.ncols() + step2 - 1)/step2;
	pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxCOMPLEX);
	auto ppar = mxGetPr(pa);
	auto ppai = mxGetPi(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j) {
			ind = i + m * j; c = a[step1*i][step2*j];
			ppar[ind] = real(c);
			ppai[ind] = imag(c);
		}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DDoub_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q, mn;
	mxArray *pa;
	m = a.dim1(); n = a.dim2(); q = a.dim3(); mn = m * n;
	m = (m + step1 - 1) / step1; n = (n + step2 - 1) / step2;
	q = (q + step3 - 1) / step3;
	size_t sz[3]{ (size_t)m,(size_t)n,(size_t)q };
	pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			for (k = 0; k < q; ++k)
				ppa[i + m * j + mn * k] = a[step1*i][step2*j][step3*k];
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}


void matsave(Mat3DDoub_I &a, const string &varname, MATFile *pfile,
			const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, mn, Nslice{ slice.size() };
	mxArray *pa;
	if (xyz == 'x') {
		Int ind;
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (j = 0; j < m; ++j)
				for (k = 0; k < n; ++k)
					ppa[j + m*k + mn*i] = a[ind][step1*j][step2*k];
		}
	}
	else if (xyz == 'y') {
		Int ind;
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (k = 0; k < m; ++k)
				for (i = 0; i < n; ++i)
					ppa[k + m*i + mn*j] = a[step2*i][ind][step1*k];
		}
	}
	else if (xyz == 'z') {
		Int ind;
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (i = 0; i < m; ++i)
				for (j = 0; j < n; ++j)
					ppa[i + m*j + mn*k] = a[step1*i][step2*j][ind];
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DComplex_I &a, const string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q, mn, ind;
	mxArray *pa;
	Complex c;
	m = a.dim1(); n = a.dim2(); q = a.dim3(); mn = m * n;
	if (step1 > 1 || step2 > 1 || step3 > 1) {
		m = (m + step1 - 1) / step1; n = (n + step2 - 1) / step2;
		q = (q + step3 - 1) / step3;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k) {
					ind = i + m * j + mn * k;
					c = a[step1*i][step2*j][step3*k];
					ppar[ind] = real(c); ppai[ind] = imag(c);
				}
	}
	else {
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k) {
					ind = i + m * j + mn * k;
					c = a[i][j][k];
					ppar[ind] = real(c); ppai[ind] = imag(c);
				}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DComplex_I &a, const string &varname, MATFile *pfile,
			const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{	
	Int i, j, k, m, n, mn, inda, Nslice{ slice.size() };
	mxArray *pa;
	Complex c;
	if (xyz == 'x') {
		Int ind;
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (j = 0; j < m; ++j)
				for (k = 0; k < n; ++k) {
					inda = j + m * k + mn * i;
					c = a[ind][step1*j][step2*k];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else if (xyz == 'y') {
		Int ind;
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (k = 0; k < m; ++k)
				for (i = 0; i < n; ++i) {
					inda = k + m * i + mn * j;
					c = a[step2*i][ind][step1*k];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else if (xyz == 'z') {
		Int ind;
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (i = 0; i < m; ++i)
				for (j = 0; j < n; ++j) {
					inda = i + m * j + mn * k;
					c = a[step1*i][step2*j][ind];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matload(Uchar &i, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	if (!mxIsUint8(ps)) {
		cout << "matload(Uchar &i...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	Uchar *pps = (Uchar *)mxGetPr(ps);
	i = pps[0];
	mxDestroyArray(ps);
}

void matload(Int &i, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	if (!mxIsInt32(ps)) {
		cout << "matload(Int &i...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	Int *pps = (Int *)mxGetPr(ps);
	i = pps[0];
	mxDestroyArray(ps);
}

void matload(Doub &s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(ps)) {
		cout << "matload(Doub &s...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	if (mxIsComplex(ps)) {
		cout << "matload(Doub &s...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	auto pps = mxGetPr(ps);
	s = pps[0];
	mxDestroyArray(ps);
}

void matload(Complex &s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(ps)) {
		cout << "matload(Complex &s...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	auto ppsr = mxGetPr(ps);
	auto ppsi = mxGetPi(ps);
	if (ppsi)
		s = Complex(ppsr[0], ppsi[0]);
	else
		s = ppsr[0];
	mxDestroyArray(ps);
}

void matload(VecUchar_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	if (!mxIsUint8(pv)) {
		cout << "matload(VecUchar_O &v...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	n = (Int)mxGetDimensions(pv)[1];
	if (v.size() != n) v.resize(n);
	Uchar *ppv = (Uchar *)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = ppv[i];
	mxDestroyArray(pv);
}

void matload(VecInt_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	if (!mxIsInt32(pv)) {
		cout << "matload(VecInt_O &v...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	n = (Int)mxGetDimensions(pv)[1];
	v.resize(n);
	Int *ppv = (Int *)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = ppv[i];
	mxDestroyArray(pv);
}

void matload(VecDoub_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(pv)) {
		cout << "matload(VecDoub_O &v...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	if (mxIsComplex(pv)) {
		cout << "matload(VecDoub_O &v...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	n = (Int)mxGetDimensions(pv)[1];
	if (v.size() != n) v.resize(n);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = ppv[i];
	mxDestroyArray(pv);
}

void matload(VecComplex_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(pv)) {
		cout << "matload(VecComplex_O &v...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	n = (Int)mxGetDimensions(pv)[1];
	v.resize(n);
	auto ppvr = mxGetPr(pv);
	auto ppvi = mxGetPi(pv);
	if (ppvi)
		for (i = 0; i < n; ++i)
			v[i] = Complex(ppvr[i], ppvi[i]);
	else
		for (i = 0; i < n; ++i)
			v[i] = ppvr[i];
	mxDestroyArray(pv);
}

void matload(MatUchar_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	if (!mxIsUint8(pa)) {
		cout << "matload(MatUchar_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	Uchar *ppa = (Uchar *)mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatInt_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	if (!mxIsInt32(pa)) {
		cout << "matload(MatInt_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	Int *ppa = (Int *)mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatDoub_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(pa)) {
		cout << "matload(MatDoub_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	if (mxIsComplex(pa)) {
		cout << "matload(MatDoub_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatComplex_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n, ind;
	mxArray *pa;
	pa = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(pa)) {
		cout << "matload(MatComplex_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppar = mxGetPr(pa);
	auto ppai = mxGetPi(pa);
	if (ppai)
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j){
			ind = i + m * j;
			a[i][j] = Complex(ppar[ind], ppai[ind]);
		}
	else
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppar[i + m*j];
	mxDestroyArray(pa);
}

void matload(Mat3DDoub_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, k, m, n, q, mn;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(pa)) {
		cout << "matload(Mat3DDoub_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	if (mxIsComplex(pa)) {
		cout << "matload(Mat3DDoub_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1]; q = (Int)sz[2]; mn = m*n;
	a.resize(m, n, q);
	auto *ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
	for (j = 0; j < n; ++j)
	for (k = 0; k < q; ++k)
		a[i][j][k] = ppa[i + m*j + mn*k];
	mxDestroyArray(pa);
}

void matload(Mat3DComplex_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, k, m, n, q, mn, ind;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	if (!mxIsDouble(pa)) {
		cout << "matload(Mat3DComplex_O &a...): wrong type!" << endl;
		exit(EXIT_FAILURE);
	}
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1]; q = (Int)sz[2]; mn = m*n;
	a.resize(m, n, q);
	auto *ppar = mxGetPr(pa);
	auto *ppai = mxGetPi(pa);
	if (ppai)
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
		for (k = 0; k < q; ++k){
			ind = i + m*j + mn*k;
			a[i][j][k] = Complex(ppar[ind], ppai[ind]);
		}
	else
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
		for (k = 0; k < q; ++k)
			a[i][j][k] = ppar[i + m*j + mn*k];
	mxDestroyArray(pa);
}

#endif

#ifndef MATFILE_BINARY

// read the next variable after previous '\n'
Int scanInverse(ifstream &fin)
{
	Char c;
	Int N;
	size_t ind, i;

	ind = fin.tellg();
	for (i = 2; i < 100; ++i) {
		fin.seekg(ind - i); c = fin.get();
		if (c == '\n') break;
	}
	fin >> N;
	fin.seekg(ind - i);
	return N;
}

// get var names and positions from the end of the file
// pfile->ind[i] points to the first matrix element;
void getprofile(MATTFile *pfile)
{
	Int j, n, temp;
	size_t i;
	vector<Int> size;
	string name;
	ifstream &fin = pfile->in;

	// read number of variables and their positions
	fin.seekg(0, fin.end);
	pfile->n = scanInverse(fin);
	for (i = 0; i < pfile->n; ++i)
		pfile->ind.push_back(scanInverse(fin));

	for (i = 0; i < pfile->n; ++i) {
		fin.seekg(pfile->ind[i]);
		// read var name
		fin >> n;
		name.resize(0);
		for (j = 0; j < n; ++j) {
			fin >> temp; name.push_back((char)temp);
		}
		pfile->name.push_back(name);
		fin >> temp; pfile->type.push_back(temp);
		fin >> n;
		size.resize(0);
		for (j = 0; j < n; ++j) {
			fin >> temp; size.push_back(temp);
		}
		pfile->size.push_back(size);
		pfile->ind[i] = fin.tellg();
	}
}

MATTFile *mattOpen(string fname, const Char *rw)
{
	// must open file in binary mode, otherwise, '\n' will be written as "\r\n"
	// and seekg() will not work the same in linux.
	#ifndef MATFILE_DUAL // TEXT mode
		fname += "t";
	#endif

	MATTFile* pfile = new MATTFile;
	if (rw[0] == 'w') {
		pfile->rw = 'w';
		pfile->n = 0;
		pfile->out = ofstream(fname, ios_base::binary);
		#ifdef MATFILE_PRECISION
			pfile->out.precision(MATFILE_PRECISION);
		#endif
	}
	else {
		pfile->rw = 'r';
		pfile->in = ifstream(fname, ios_base::binary);
		if (!pfile->in) {
			cout << "error: file not found: " << fname << endl;
			exit(EXIT_FAILURE);
		}
		pfile->in.precision(17);
		getprofile(pfile); // get var names
	}
	return pfile;
}

void mattClose(MATTFile* pfile)
{
	Int i;
	if (pfile->rw == 'w') {
		ofstream &fout = pfile->out;
		// write position of variables
		for (i = (Int)pfile->ind.size() - 1; i >= 0; --i)
			fout << pfile->ind[i] << "\n";
		// write number of variables
		fout << pfile->n;
		pfile->out.close();
	}
	else {
		pfile->in.close();
	}
	delete pfile;
}

void mattsave(const Uchar s, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 3 << '\n';
	// write dimension info
	fout << 0 << '\n';
	// write matrix data
	fout << (Int)s << '\n';
}

void mattsave(const Int s, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 2 << '\n';
	// write dimension info
	fout << 0 << '\n';
	// write matrix data
	fout << s << '\n';
}

void mattsave(const Doub s, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 0 << '\n';
	// write dimension info
	fout << 0 << '\n';
	// write matrix data
	fout << s << '\n';
}

void mattsave(const Complex s, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 1 << '\n';
	// write dimension info
	fout << 0 << '\n';
	// write matrix data
	if (imag(s) == 0)
		fout << real(s) << '\n';
	else if (imag(s) < 0)
		fout << real(s) << imag(s) << "i\n";
	else
		fout << real(s) << '+' << imag(s) << "i\n";
}

void mattsave(VecUchar_I &v, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 3 << '\n';
	// write dimension info
	n = v.size();
	fout << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		fout << (Int)v[i] << '\n';
	}
}

void mattsave(VecInt_I &v, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 2 << '\n';
	// write dimension info
	n = v.size();
	fout << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		fout << v[i] << '\n';
	}
}

void mattsave(VecDoub_I &v, const string &varname, MATTFile *pfile)
{
	Int i, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 0 << '\n';
	// write dimension info
	n = v.size();
	fout << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		fout << v[i] << '\n';
	}
}

void mattsave(VecComplex_I &v, const string &varname, MATTFile *pfile)
{
	Int i, n;
	Doub cr, ci;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 1 << '\n';
	// write dimension info
	n = v.size();
	fout << 1 << '\n' << n << '\n';
	// write matrix data
	for (i = 0; i < n; ++i) {
		cr = real(v[i]); ci = imag(v[i]);
		if (ci == 0)
			fout << cr << '\n';
		else if (ci < 0)
			fout << cr << ci << "i\n";
		else
			fout << cr << '+' << ci << "i\n";
	}
}

void mattsave(MatUchar_I &a, const string &varname, MATTFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 3 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	fout << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			fout << (Int)a[step1*i][step2*j] << '\n';
		}
}

void mattsave(MatInt_I &a, const string &varname, MATTFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 2 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	fout << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			fout << a[step1*i][step2*j] << '\n';
		}
}

void mattsave(MatDoub_I &a, const string &varname, MATTFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 0 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	fout << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			fout << a[step1*i][step2*j] << '\n';
		}
}

void mattsave(MatComplex_I &a, const string &varname, MATTFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	Complex c; Doub cr, ci;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 1 << '\n';
	// write dimension info
	m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
	fout << 2 << '\n' << m << '\n' << n << '\n';
	// write matrix data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			c = a[step1*i][step2*j]; cr = real(c); ci = imag(c);
			if (ci == 0)
				fout << cr << '\n';
			else if (ci < 0)
				fout << cr << ci << "i\n";
			else
				fout << cr << '+' << ci << "i\n";
		}
}

void mattsave(Mat3DDoub_I &a, const string &varname, MATTFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 0 << '\n';
	// write dimension info
	m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
	q = (a.dim3() + step3 - 1) / step3;
	fout << 3 << '\n' << m << '\n' << n << '\n' << q << '\n';
	// write matrix data
	for (k = 0; k < q; ++k)
	for (j = 0; j < n; ++j)
	for (i = 0; i < m; ++i)
		fout << a[step1*i][step2*j][step3*k] << '\n';
}

void mattsave(Mat3DDoub_I &a, const string &varname, MATTFile *pfile,
	const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, ind, Nslice{ slice.size() };
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 0 << '\n';
	if (xyz == 'x') {
		// write dimension info
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2;
		fout << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (k = 0; k < n; ++k)
				for (j = 0; j < m; ++j)
					fout << a[ind][step1*j][step2*k] << '\n';
		}
	}
	else if (xyz == 'y') {
		// write dimension info
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2;
		fout << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (i = 0; i < n; ++i)
				for (k = 0; k < m; ++k)
					fout << a[step2*i][ind][step1*k] << '\n';
		}
	}
	else if (xyz == 'z') {
		// write dimension info
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
		fout << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (j = 0; j < n; ++j)
				for (i = 0; i < m; ++i)
					fout << a[step1*i][step2*j][ind] << '\n';
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
}

void mattsave(Mat3DComplex_I &a, const string &varname, MATTFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q;
	Complex c; Doub cr, ci;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 1 << '\n';
	// write dimension info
	m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
	q = (a.dim3() + step3 - 1) / step3;
	fout << 3 << '\n' << m << '\n' << n << '\n' << q << '\n';
	// write matrix data
	for (k = 0; k < q; ++k)
		for (j = 0; j < n; ++j)
			for (i = 0; i < m; ++i) {
				c = a[step1*i][step2*j][step3*k]; cr = real(c); ci = imag(c);
				if (ci == 0)
					fout << cr << '\n';
				else if (ci < 0)
					fout << cr << ci << "i\n";
				else
					fout << cr << '+' << ci << "i\n";
			}
}

void mattsave(Mat3DComplex_I &a, const string &varname, MATTFile *pfile,
	const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, ind, Nslice{ slice.size() };
	Complex c; Doub cr, ci;
	ofstream &fout = pfile->out;
	++pfile->n; pfile->ind.push_back(fout.tellp());
	// write variable name info
	n = (Int)varname.size();
	fout << n << '\n';
	for (i = 0; i < n; ++i) {
		fout << (Int)varname.at(i) << '\n';
	}
	// write data type info
	fout << 1 << '\n';
	if (xyz == 'x') {
		// write dimension info
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2;
		fout << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (k = 0; k < n; ++k)
				for (j = 0; j < m; ++j)
				{
					c = a[ind][step1*j][step2*k]; cr = real(c); ci = imag(c);
					if (ci == 0)
						fout << cr << '\n';
					else if (ci < 0)
						fout << cr << ci << "i\n";
					else
						fout << cr << '+' << ci << "i\n";
				}
					
		}
	}
	else if (xyz == 'y') {
		// write dimension info
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2;
		fout << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (i = 0; i < n; ++i)
				for (k = 0; k < m; ++k) {
					c = a[step2*i][ind][step1*k]; cr = real(c); ci = imag(c);
					if (ci == 0)
						fout << cr << '\n';
					else if (ci < 0)
						fout << cr << ci << "i\n";
					else
						fout << cr << '+' << ci << "i\n";
				}
		}
	}
	else if (xyz == 'z') {
		// write dimension info
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2;
		fout << 3 << '\n' << m << '\n' << n << '\n' << Nslice << '\n';
		// write matrix data
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (j = 0; j < n; ++j)
				for (i = 0; i < m; ++i) {
					c = a[step1*i][step2*j][ind]; cr = real(c); ci = imag(c);
					if (ci == 0)
						fout << cr << '\n';
					else if (ci < 0)
						fout << cr << ci << "i\n";
					else
						fout << cr << '+' << ci << "i\n";
				}
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
}

// search variable in file by name
inline Int nameSearch(const string &name, MATTFile *pfile)
{
	for (Int i = 0; i < pfile->n; ++i)
		if (name == pfile->name[i])
			return i;
	cout << "\n\n error:variable name not found! line: " << __LINE__ << endl;
	return -1; // did not fine
}

inline void scanComplex(Complex &c, ifstream &fin)
{
	Doub cr = 0, ci = 0;
	Uchar ch;
	fin >> cr;
	ch = fin.get();
	if (ch == '\n') {
		c = cr; return;
	}
	fin >> ci;
	if (ch == '-')
		ci *= -1.;
	c = Complex(cr, ci);
	fin.ignore(100, '\n');
}

void mattload(Uchar &I, const string &varname, MATTFile *pfile)
{
	Int i, temp;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	if (pfile->type[i] != 3 || pfile->size[i].size() != 0) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	// read var data
	fin >> temp; I = Uchar(temp);
}

void mattload(Int &I, const string &varname, MATTFile *pfile)
{
	Int i;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);
	
	// read var type and dim
	if (pfile->type[i] < 2 || pfile->size[i].size() != 0) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	// read var data
	fin >> I;
}

void mattload(Doub &I, const string &varname, MATTFile *pfile)
{
	Int i;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	if (pfile->type[i] == 1 || pfile->size[i].size() != 0) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	// read var data
	fin >> I;
}

void mattload(Complex &I, const string &varname, MATTFile *pfile)
{
	Int i;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	if (pfile->size[i].size() != 0) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	// read var data
	scanComplex(I, fin);
}

void mattload(VecUchar_O &v, const string &varname, MATTFile *pfile)
{
	Int i, dim, n, temp;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] != 3 || dim != 1) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	n = pfile->size[i][0]; v.resize(n);
	// read var data
	for (i = 0; i < n; ++i) {
		fin >> temp;  v[i] = (Uchar)temp;
	}
}

void mattload(VecInt_O &v, const string &varname, MATTFile *pfile)
{
	Int i, dim, n;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] < 2 || dim != 1) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	n = pfile->size[i][0]; v.resize(n);
	// read var data
	for (i = 0; i < n; ++i)
		fin >> v[i];
}

void mattload(VecDoub_O &v, const string &varname, MATTFile *pfile)
{
	Int i, dim, n;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] == 1 || dim != 1) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	n = pfile->size[i][0]; v.resize(n);
	// read var data
	for (i = 0; i < n; ++i)
		fin >> v[i];
}

void mattload(VecComplex_O &v, const string &varname, MATTFile *pfile)
{
	Int i, dim, n;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (dim != 1) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	n = pfile->size[i][0]; v.resize(n);
	// read var data
	for (i = 0; i < n; ++i)
		scanComplex(v[i], fin);
}

void mattload(MatUchar_O &a, const string &varname, MATTFile *pfile)
{
	Int i, j, dim, m, n, temp;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] != 3 || dim != 2) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	m = pfile->size[i][0]; n = pfile->size[i][1]; a.resize(m, n);
	// read var data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i) {
			fin >> temp;  a[i][j] = (Uchar)temp;
		}
}

void mattload(MatInt_O &a, const string &varname, MATTFile *pfile)
{
	Int i, j, dim, m, n;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] < 2 || dim != 2) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	m = pfile->size[i][0]; n = pfile->size[i][1]; a.resize(m, n);
	// read var data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i)
			fin >> a[i][j];
}

void mattload(MatDoub_O &a, const string &varname, MATTFile *pfile)
{
	Int i, j, dim, m, n;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] == 1 || dim != 2) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	m = pfile->size[i][0]; n = pfile->size[i][1]; a.resize(m, n);
	// read var data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i)
			fin >> a[i][j];
}

void mattload(MatComplex_O &a, const string &varname, MATTFile *pfile)
{
	Int i, j, dim, m, n;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (dim != 2) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	m = pfile->size[i][0]; n = pfile->size[i][1]; a.resize(m, n);
	// read var data
	for (j = 0; j < n; ++j)
		for (i = 0; i < m; ++i)
			scanComplex(a[i][j], fin);
}

void mattload(Mat3DDoub_O &a, const string &varname, MATTFile *pfile)
{
	Int i, j, k, dim, m, n, q;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (pfile->type[i] == 1 || dim != 3) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	m = pfile->size[i][0]; n = pfile->size[i][1]; q = pfile->size[i][2];
	a.resize(m, n, q);
	// read var data
	for (k = 0; k < q; ++k)
		for (j = 0; j < n; ++j)
			for (i = 0; i < m; ++i)
				fin >> a[i][j][k];
}

void mattload(Mat3DComplex_O &a, const string &varname, MATTFile *pfile)
{
	Int i, j, k, dim, m, n, q;
	ifstream &fin = pfile->in;
	i = nameSearch(varname, pfile);
	fin.seekg(pfile->ind[i]);

	// read var type and dim
	dim = (Int)pfile->size[i].size();
	if (dim != 3) {
		cout << "\n\n error: wrong type or dim! line: " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
	m = pfile->size[i][0]; n = pfile->size[i][1]; q = pfile->size[i][2];
	a.resize(m, n, q);
	// read var data
	for (k = 0; k < q; ++k)
		for (j = 0; j < n; ++j)
			for (i = 0; i < m; ++i)
				scanComplex(a[i][j][k], fin);
}

#endif

#ifdef MATFILE_DUAL
void mat2matt(const string &fmat, const string &fmatt)
{
	Int i, n, ndim;
	char **names;
	MATFile *pfmat = matOpen(fmat.c_str(), "r");
	MATTFile *pfmatt = mattOpen(fmatt, "w");
	
	// get number of variables n and names
	names = matGetDir(pfmat, &n);
	for (i = 0; i < n; ++i){
		mxArray *pa = matGetVariable(pfmat, names[i]);
		ndim = (Int)mxGetNumberOfDimensions(pa);
		const mwSize *sz = mxGetDimensions(pa);
		if (ndim == 2) {
			if (sz[0] == 0 || sz[1] == 0) {
				cout << "\n\nerror: empty variable unsupported!" << __LINE__ << endl;
				exit(EXIT_FAILURE);
			}
			else if (sz[0] == 1 && sz[1] == 1) {
				// scalar
				if (mxIsComplex(pa)) {
					Complex c;
					matload(c, names[i], pfmat);
					mattsave(c, names[i], pfmatt);
				}
				else if (mxIsDouble(pa)) {
					Doub s;
					matload(s, names[i], pfmat);
					mattsave(s, names[i], pfmatt);
				}
				else if (mxIsInt32(pa)) {
					Int s;
					matload(s, names[i], pfmat);
					mattsave(s, names[i], pfmatt);
				}
				else if (mxIsUint8(pa)) {
					Uchar ch;
					matload(ch, names[i], pfmat);
					mattsave(ch, names[i], pfmatt);
				}
				else {
					cout << "\n\nerror: type unsupported!" << __LINE__ << endl;
					exit(EXIT_FAILURE);
				}
			}
			else if (sz[0] == 1 || sz[1] == 1) {
				// save to vector
				if (mxIsComplex(pa)) {
					VecComplex v;
					matload(v, names[i], pfmat);
					mattsave(v, names[i], pfmatt);
				}
				else if (mxIsDouble(pa)) {
					VecDoub v;
					matload(v, names[i], pfmat);
					mattsave(v, names[i], pfmatt);
				}
				else if (mxIsInt32(pa)) {
					VecInt v;
					matload(v, names[i], pfmat);
					mattsave(v, names[i], pfmatt);
				}
				else if (mxIsUint8(pa)) {
					VecUchar v;
					matload(v, names[i], pfmat);
					mattsave(v, names[i], pfmatt);
				}
				else {
					cout << "\n\nerror: type unsupported!" << __LINE__ << endl;
					exit(EXIT_FAILURE);
				}
			}
			else {
				// save to matrix
				if (mxIsComplex(pa)) {
					MatComplex a;
					matload(a, names[i], pfmat);
					mattsave(a, names[i], pfmatt);
				}
				else if (mxIsDouble(pa)) {
					MatDoub a;
					matload(a, names[i], pfmat);
					mattsave(a, names[i], pfmatt);
				}
				else if (mxIsInt32(pa)) {
					MatInt a;
					matload(a, names[i], pfmat);
					mattsave(a, names[i], pfmatt);
				}
				else if (mxIsUint8(pa)) {
					MatUchar a;
					matload(a, names[i], pfmat);
					mattsave(a, names[i], pfmatt);
				}
				else {
					cout << "\n\nerror: type unsupported!" << __LINE__ << endl;
					exit(EXIT_FAILURE);
				}
			}
		}
		else if (ndim == 3) {
			// save to matrix
			if (mxIsComplex(pa)) {
				Mat3DComplex a3;
				matload(a3, names[i], pfmat);
				mattsave(a3, names[i], pfmatt);
			}
			else if (mxIsDouble(pa)) {
				Mat3DDoub a3;
				matload(a3, names[i], pfmat);
				mattsave(a3, names[i], pfmatt);
			}
			else {
				cout << "\n\nerror: type unsupported!" << __LINE__ << endl;
				exit(EXIT_FAILURE);
			}
		}
		else {
			cout << "\n\n error: unsupported dimension! " << __LINE__ << endl;
			exit(EXIT_FAILURE);
		}
	}
	matClose(pfmat);
	mattClose(pfmatt);
}

void matt2mat(const string &fmatt, const string &fmat)
{

}
#endif