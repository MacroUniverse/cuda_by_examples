#include "hip/hip_runtime.h"
// save vectors and matrices defined in "nr3.h" (see Numerical Recipes) into .mat file for Matlab
// can't figure out how to directly save to complex matrix yet

#include "matsave.h"

using namespace std;

// matsave()

void matsave(const Uchar s, const std::string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateNumericMatrix(1, 1, mxUINT8_CLASS, mxREAL);
	Uchar *pps = (Uchar*)mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Int s, const std::string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
	Int *pps = (Int*)mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Doub s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = mxCreateDoubleMatrix(1, 1, mxREAL);
	auto pps = mxGetPr(ps);
	pps[0] = s;
	matPutVariable(pfile, varname.c_str(), ps);
	mxDestroyArray(ps);
}

void matsave(const Complex s, const string &varname, MATFile *pfile)
{
	mxArray *pa;
	pa = mxCreateDoubleMatrix(1, 1, mxCOMPLEX);

	auto ppa = mxGetPr(pa);
	ppa[0] = real(s);
	ppa = mxGetPi(pa);
	ppa[0] = imag(s);
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(VecUchar_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxUINT8_CLASS, mxREAL);
	Uchar *ppv = (Uchar*)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecInt_I &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxINT32_CLASS, mxREAL);
	Int *ppv = (Int*)mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecDoub_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxDOUBLE_CLASS, mxREAL);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		ppv[i] = v[i];
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(VecComplex_I &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	string str;
	mxArray *pv;
	n = v.size();
	pv = mxCreateUninitNumericMatrix(1, n, mxDOUBLE_CLASS, mxCOMPLEX);
	auto ppvr = mxGetPr(pv);
	auto ppvi = mxGetPi(pv);
	for (i = 0; i < n; ++i) {
		ppvr[i] = real(v[i]);
		ppvi[i] = imag(v[i]);
	}
	matPutVariable(pfile, varname.c_str(), pv);
	mxDestroyArray(pv);
}

void matsave(MatUchar_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxUINT8_CLASS, mxREAL);
		Uchar *ppa = (Uchar*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[step1*i][step2*j];
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxUINT8_CLASS, mxREAL);
		Uchar *ppa = (Uchar*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[i][j];
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatInt_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxINT32_CLASS, mxREAL);
		Int *ppa = (Int*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[step1*i][step2*j];
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxINT32_CLASS, mxREAL);
		Int *ppa = (Int*)mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m * j] = a[i][j];
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatDoub_I &a, const string &varname, MATFile *pfile, 
			const Int step1, const Int step2)
{
	Int i, j, m, n;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1) / step1; n = (a.ncols() + step2 - 1) / step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m*j] = a[step1*i][step2*j];
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ppa[i + m*j] = a[i][j];
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(MatComplex_I &a, const string &varname, MATFile *pfile,
			const Int step1, const Int step2)
{
	Int i, j, m, n, ind;
	Complex c;
	mxArray *pa;
	if (step1 > 1 || step2 > 1) {
		m = (a.nrows() + step1 - 1)/step1; n = (a.ncols() + step2 - 1)/step2;
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa);
		auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ind = i + m * j; c = a[step1*i][step2*j];
				ppar[ind] = real(c);
				ppai[ind] = imag(c);
			}
	}
	else {
		m = a.nrows(); n = a.ncols();
		pa = mxCreateUninitNumericMatrix(m, n, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa);
		auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j) {
				ind = i + m * j; c = a[i][j];
				ppar[ind] = real(c);
				ppai[ind] = imag(c);
			}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DDoub_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q, mn;
	mxArray *pa;
	m = a.dim1(); n = a.dim2(); q = a.dim3(); mn = m * n;
	if (step1 > 1 || step2 > 1 || step3 > 1) {
		m = (m + step1 - 1) / step1; n = (n + step2 - 1) / step2;
		q = (q + step3 - 1) / step3;
		size_t sz[3]{ (size_t)m,(size_t)n,(size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k)
					ppa[i + m * j + mn * k] = a[step1*i][step2*j][step3*k];
	}
	else {
		size_t sz[3]{ (size_t)m,(size_t)n,(size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k)
					ppa[i + m * j + mn * k] = a[i][j][k];
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

/* specify xyz = 'x','y' or 'z', and take Nslice at indslice[i]
if xyz = 'x', step1 is in y direction, step2 is in z direction, save pa[iy][iz][ix].
if xyz = 'y', step1 is in z direction, step2 is in x direction, save pa[iz][ix][iy].
if xyz = 'z', step1 is in x direction, step2 is in y direction, save pa[ix][iy][iz]. */
void matsave(Mat3DDoub_I &a, const std::string &varname, MATFile *pfile,
			const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{
	Int i, j, k, m, n, mn, Nslice{ slice.size() };
	mxArray *pa;
	if (xyz == 'x') {
		Int ind;
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (j = 0; j < m; ++j)
				for (k = 0; k < n; ++k)
					ppa[j + m*k + mn*i] = a[ind][step1*j][step2*k];
		}
	}
	else if (xyz == 'y') {
		Int ind;
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (k = 0; k < m; ++k)
				for (i = 0; i < n; ++i)
					ppa[k + m*i + mn*j] = a[step2*i][ind][step1*k];
		}
	}
	else if (xyz == 'z') {
		Int ind;
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxREAL);
		auto ppa = mxGetPr(pa);
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (i = 0; i < m; ++i)
				for (j = 0; j < n; ++j)
					ppa[i + m*j + mn*k] = a[step1*i][step2*j][ind];
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

void matsave(Mat3DComplex_I &a, const std::string &varname, MATFile *pfile,
	const Int step1, const Int step2, const Int step3)
{
	Int i, j, k, m, n, q, mn, ind;
	mxArray *pa;
	Complex c;
	m = a.dim1(); n = a.dim2(); q = a.dim3(); mn = m * n;
	if (step1 > 1 || step2 > 1 || step3 > 1) {
		m = (m + step1 - 1) / step1; n = (n + step2 - 1) / step2;
		q = (q + step3 - 1) / step3;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k) {
					ind = i + m * j + mn * k;
					c = a[step1*i][step2*j][step3*k];
					ppar[ind] = real(c); ppai[ind] = imag(c);
				}
	}
	else {
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)q };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < m; ++i)
			for (j = 0; j < n; ++j)
				for (k = 0; k < q; ++k) {
					ind = i + m * j + mn * k;
					c = a[i][j][k];
					ppar[ind] = real(c); ppai[ind] = imag(c);
				}
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

/* specify xyz = 'x','y' or 'z', and take Nslice at indslice[i]
if xyz = 'x', step1 is in y direction, step2 is in z direction, save pa[iy][iz][ix].
if xyz = 'y', step1 is in z direction, step2 is in x direction, save pa[iz][ix][iy].
if xyz = 'z', step1 is in x direction, step2 is in y direction, save pa[ix][iy][iz]. */
void matsave(Mat3DComplex_I &a, const std::string &varname, MATFile *pfile,
			const Char xyz, VecInt_I &slice, const Int step1, const Int step2)
{	
	Int i, j, k, m, n, mn, inda, Nslice{ slice.size() };
	mxArray *pa;
	Complex c;
	if (xyz == 'x') {
		Int ind;
		m = (a.dim2() + step1 - 1) / step1; n = (a.dim3() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (i = 0; i < Nslice; ++i) {
			ind = slice[i];
			for (j = 0; j < m; ++j)
				for (k = 0; k < n; ++k) {
					inda = j + m * k + mn * i;
					c = a[ind][step1*j][step2*k];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else if (xyz == 'y') {
		Int ind;
		m = (a.dim3() + step1 - 1) / step1; n = (a.dim1() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (j = 0; j < Nslice; ++j) {
			ind = slice[j];
			for (k = 0; k < m; ++k)
				for (i = 0; i < n; ++i) {
					inda = k + m * i + mn * j;
					c = a[step2*i][ind][step1*k];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else if (xyz == 'z') {
		Int ind;
		m = (a.dim1() + step1 - 1) / step1; n = (a.dim2() + step2 - 1) / step2; mn = m * n;
		size_t sz[3]{ (size_t)m, (size_t)n, (size_t)Nslice };
		pa = mxCreateUninitNumericArray(3, sz, mxDOUBLE_CLASS, mxCOMPLEX);
		auto ppar = mxGetPr(pa); auto ppai = mxGetPi(pa);
		for (k = 0; k < Nslice; ++k) {
			ind = slice[k];
			for (i = 0; i < m; ++i)
				for (j = 0; j < n; ++j) {
					inda = i + m * j + mn * k;
					c = a[step1*i][step2*j][ind];
					ppar[inda] = real(c); ppai[inda] = imag(c);
				}
		}
	}
	else {
		cout << "error! illegal value of xyz" << endl; return;
	}
	matPutVariable(pfile, varname.c_str(), pa);
	mxDestroyArray(pa);
}

// matload()

void matload(Int &i, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	auto pps = mxGetPr(ps);
	i = (Int)pps[0];
	mxDestroyArray(ps);
}

void matload(Doub &s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	auto pps = mxGetPr(ps);
	s = pps[0];
	mxDestroyArray(ps);
}

void matload(Complex &s, const string &varname, MATFile *pfile)
{
	mxArray *ps;
	ps = matGetVariable(pfile, varname.c_str());
	auto ppsr = mxGetPr(ps);
	auto ppsi = mxGetPi(ps);
	if (ppsi)
		s = Complex(ppsr[0], ppsi[0]);
	else
		s = ppsr[0];
	mxDestroyArray(ps);
}

void matload(VecInt_O &v, const std::string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	n = (Int)mxGetDimensions(pv)[1];
	if (v.size() != n) v.resize(n);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = (Int)round(ppv[i]);
	mxDestroyArray(pv);
}

void matload(VecDoub_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	n = (Int)mxGetDimensions(pv)[1];
	if (v.size() != n) v.resize(n);
	auto ppv = mxGetPr(pv);
	for (i = 0; i < n; ++i)
		v[i] = ppv[i];
	mxDestroyArray(pv);
}

void matload(VecComplex_O &v, const string &varname, MATFile *pfile)
{
	Int i, n;
	mxArray *pv;
	pv = matGetVariable(pfile, varname.c_str());
	n = (Int)mxGetDimensions(pv)[1];
	v.resize(n);
	auto ppvr = mxGetPr(pv);
	auto ppvi = mxGetPi(pv);
	if (ppvi)
		for (i = 0; i < n; ++i)
			v[i] = Complex(ppvr[i], ppvi[i]);
	else
		for (i = 0; i < n; ++i)
			v[i] = ppvr[i];
	mxDestroyArray(pv);
}

void matload(MatInt_O &a, const std::string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = (Int)ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatDoub_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppa[m*j + i];
	mxDestroyArray(pa);
}

void matload(MatComplex_O &a, const string &varname, MATFile *pfile)
{
	Int i, j, m, n, ind;
	mxArray *pa;
	pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1];
	a.resize(m, n);
	auto ppar = mxGetPr(pa);
	auto ppai = mxGetPi(pa);
	if (ppai)
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j){
			ind = i + m * j;
			a[i][j] = Complex(ppar[ind], ppai[ind]);
		}
	else
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			a[i][j] = ppar[i + m*j];
	mxDestroyArray(pa);
}

void matload(Mat3DDoub_O &a, const std::string &varname, MATFile *pfile)
{
	Int i, j, k, m, n, q, mn;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1]; q = (Int)sz[2]; mn = m*n;
	a.resize(m, n, q);
	auto *ppa = mxGetPr(pa);
	for (i = 0; i < m; ++i)
	for (j = 0; j < n; ++j)
	for (k = 0; k < q; ++k)
		a[i][j][k] = ppa[i + m*j + mn*k];
	mxDestroyArray(pa);
}

void matload(Mat3DComplex_O &a, const std::string &varname, MATFile *pfile)
{
	Int i, j, k, m, n, q, mn, ind;
	mxArray *pa = matGetVariable(pfile, varname.c_str());
	const mwSize *sz = mxGetDimensions(pa);
	m = (Int)sz[0]; n = (Int)sz[1]; q = (Int)sz[2]; mn = m*n;
	a.resize(m, n, q);
	auto *ppar = mxGetPr(pa);
	auto *ppai = mxGetPi(pa);
	if (ppai)
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
		for (k = 0; k < q; ++k){
			ind = i + m*j + mn*k;
			a[i][j][k] = Complex(ppar[ind], ppai[ind]);
		}
	else
		for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
		for (k = 0; k < q; ++k)
			a[i][j][k] = ppar[i + m*j + mn*k];
	mxDestroyArray(pa);
}
