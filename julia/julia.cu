#include "hip/hip_runtime.h"
#ifdef _MSC_VER
#include "hip/hip_runtime.h"
#include ""
#endif

#include "matsave.h"
#include "nr3plus.h"
using namespace std;
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost

__device__
Int julia(Int x, Int y, float scale, Int dim) {
	Int i;
	float jx = scale * (float)(x - dim/2)/(dim/2);
	float jy = scale * (float)(y - dim/2)/(dim/2);
	Complex c(-0.8, 0.15745);
	Complex a(jx, jy);

	for (i=0; i<200; i++) {
		a = a * a + c;
		if (abs(a) > 31.62)
			return 0;
	}
	return 1;
}

__global__
void kernel(Uchar *ptr, float scale, Int dim) {
	Int indxy, ind = blockIdx.x*blockDim.x + threadIdx.x;
	Int stride = blockDim.x*gridDim.x;
	Int x,y;
	for(indxy = ind; indxy < dim*dim; indxy += stride){
		x = indxy%dim; y = indxy/dim;
		Int juliaValue = julia( x, y, scale, dim);
		ptr[indxy] = (Uchar)(255*juliaValue);
	}
}

int main( void ) {
	Int dim = 1000;
	int i, img_size = dim*dim*sizeof(Uchar);
	float scale = 1.5;
	string str;
	MatUchar bitmap(dim, dim);
	Uchar *dev_bitmap;

	hipMalloc( (void**)&dev_bitmap, img_size );

	for (i = 0; i < 150; ++i){
		scale *= 0.95;
		cout << "iteration: " << i << endl;
		kernel<<<320,32>>>(dev_bitmap, scale, dim);
		hipMemcpy(bitmap[0], dev_bitmap, img_size, D2H);
		str = to_string(i);
		MATFile *pfile = matOpen(("Data/" + str + ".julia.mat").c_str(), "w");
		matsave(bitmap, "I", pfile);
		matClose(pfile);
	}

	hipFree( dev_bitmap );
}
